#include "hip/hip_runtime.h"
/***************************************************************************
# Copyright (c) 2019, NVIDIA CORPORATION. All rights reserved.
#
# Redistribution and use in source and binary forms, with or without
# modification, are permitted provided that the following conditions
# are met:
#  * Redistributions of source code must retain the above copyright
#    notice, this list of conditions and the following disclaimer.
#  * Redistributions in binary form must reproduce the above copyright
#    notice, this list of conditions and the following disclaimer in the
#    documentation and/or other materials provided with the distribution.
#  * Neither the name of NVIDIA CORPORATION nor the names of its
#    contributors may be used to endorse or promote products derived
#    from this software without specific prior written permission.
#
# THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
# EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
# IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
# PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
# CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
# EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
# PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
# PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
# OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
# (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
# OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
***************************************************************************/
#include "CopySurface.h"
#include <>

// The CUDA kernel. This sample simply copies the input surface.
template<class T>
__global__ void copySurface(hipSurfaceObject_t input, hipSurfaceObject_t output, unsigned int width, unsigned int height)
{
    unsigned int x = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int y = blockIdx.y * blockDim.y + threadIdx.y;
    if (x < width && y < height)
    {
        T data;
        surf2Dread(&data, input, sizeof(T) * x, y);
        surf2Dwrite(data, output, sizeof(T) * x, y);
    }
}

// A wrapper function that launches the kernel.
void launchCopySurface(hipSurfaceObject_t input, hipSurfaceObject_t output, unsigned int width, unsigned int height, unsigned int format)
{
    dim3 dimBlock(16, 16);
    dim3 dimGrid((width + dimBlock.x - 1) / dimBlock.x, (height + dimBlock.y - 1) / dimBlock.y);
    if (format == hipChannelFormatKindFloat) copySurface<float><<<dimGrid, dimBlock>>>(input, output, width, height);
    else copySurface<int><<<dimGrid, dimBlock>>>(input, output, width, height);
}
